#include "hip/hip_runtime.h"
/*
 * cudaFuncs.cu
 *  Created on: Jul 18, 2021
 *      Author: Shahar Hikri
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#define CUDA_LOGIC
#include "FinalProj.h"

/*same as strlen func (from 'string.h').
*/
__host__ __device__ int strlen_dh(const char* str){
	int i=0;
	while(str[i]!='\0')
		i++;
	return i;
}

/*if str contains c return true
  else return false.
*/
__host__ __device__ bool isContains(const char* str, char c){
	if(str==NULL)
		return false;
	
	int str_len = strlen_dh(str);
	int i=0;
	
	for(i=0;i<str_len;i++){
		if(str[i]==c)
			return true;
	}
	return false;
}

/*if c2 in c2In_str: leave it's val and return.
 * else: replace C2 with char in c2In_str but not in c2NotIn_str.*/
__host__ __device__  bool replaceC2(char c1,char* c2_p,const char* c2In_str,const char* c2NotIn_str){
	char c2 = *c2_p;

	if(c1<'A' || c1>'Z' || c2<'A' || c2>'Z'){
		return false;
	}

	if(isContains(c2In_str, c2)) //leave c2 as is
		return true;

	int c2In_str_len = strlen_dh(c2In_str);
	int i;
	for(i=0;i<c2In_str_len;i++){
		if((c2NotIn_str==NULL || !isContains(c2NotIn_str, c2In_str[i])) ){
			*c2_p = c2In_str[i];
			return true;
		}
	}
	return false;
}

/*replacing c2 to optimal value and return the score that c1 and the new c2 gives
  - won't change c2 value if it cant replace its value to somthing that gives better score with c1.
  - semi&con priority:  for example if semi weight better then CON wieght, the function will try to change c2(if not already is) 
						to semi that not con with c1, if it didn't find semi and not con, it will try to give c2 semi that is also con with c1.
						In either case it will bring back semi's weight.
*/
__host__ __device__ double replaceC2toOpt(char c1,char* c2_p, double* W , int min_max_flag, int* WindArr){
	#ifdef  __CUDA_ARCH__ //gpu(device) using this function
	
	#define CONANDSEMI_DICT ConAndSemi_dict_d
	#define SEMIANDNOTCON_DICT SemiAndNotCon_dict_d
	#define CONANDNOTSEMI_DICT ConAndSemi_dict_d
	#define CON_DICT Con_dict_d
	#define NOTCONANDNOTSEMI_DICT notConAndNotSemi_dict_d

	#else  //cpu(host) using this function

	#define CONANDSEMI_DICT ConAndSemi_dict
	#define SEMIANDNOTCON_DICT SemiAndNotCon_dict
	#define CONANDNOTSEMI_DICT ConAndSemi_dict
	#define CON_DICT Con_dict
	#define NOTCONANDNOTSEMI_DICT notConAndNotSemi_dict

	#endif

	char c2 = *c2_p;

	int i;

	bool searched_in_ConAndSemi_flag = false;

	for(i=0;i<W_LEN;i++){

		if(WindArr[i]==DOT){
			if(replaceC2(c1, c2_p, SEMIANDNOTCON_DICT[c1-'A'], CON_DICT[c2-'A']) )
				return W[DOT];
			else if(!searched_in_ConAndSemi_flag && replaceC2(c1, c2_p, CONANDSEMI_DICT[c1-'A'], CON_DICT[c2-'A']))
				return W[DOT];
			else
				searched_in_ConAndSemi_flag=true;
		}

		else if(WindArr[i]==COL){

			if(replaceC2(c1, c2_p, CONANDNOTSEMI_DICT[c1-'A'], CON_DICT[c2-'A']) )
				return W[COL];
			else if(!searched_in_ConAndSemi_flag && replaceC2(c1, c2_p, CONANDSEMI_DICT[c1-'A'], CON_DICT[c2-'A']))
				return W[COL];
			else
				searched_in_ConAndSemi_flag=true;
		}

		else if(WindArr[i]==AST){
			char c2In_str[2] = {c1 , '\0' };
			if(replaceC2(c1, c2_p, c2In_str, CON_DICT[c2-'A']))
				return W[AST];
		}

		else{
			if(replaceC2(c1, c2_p, NOTCONANDNOTSEMI_DICT[c1-'A'], CON_DICT[c2-'A']))
				return W[SPC];
		}
	}
	return 0;
} 


__global__ void fillDict(char* proc_seq1,char* proc_seq2, double* W, int* WindArr, int min_max_flag, double* proc_sum_p){
	int i = threadIdx.x;
	if(i==0)
		*proc_sum_p = 0;
	__syncthreads();
	
	double theadWeight = replaceC2toOpt(proc_seq1[i],(proc_seq2+i),W,min_max_flag,WindArr);

	int j;
	for(j=0;j<26;j++){
		if(i==j)
			(*proc_sum_p)+=theadWeight;
			
		__syncthreads();
	}
}

double computeOnGPU(char *proc_seq1,char *proc_seq2, int seq_len, double W[4], int min_max_flag, int WindArr[4]) {

	char *proc_seq1_d, *proc_seq2_d;
	double* cudaWeight_p_d;
	double cudaWeight = 0;
	double *W_d;
	int* WindArr_d;
	
	hipMalloc((void **)&proc_seq1_d, seq_len*sizeof(char));
	hipMalloc((void **)&proc_seq2_d, seq_len*sizeof(char));
	hipMalloc((void **)&cudaWeight_p_d, sizeof(double));
	hipMalloc((void **)&W_d, 4*sizeof(double));
	hipMalloc((void **)&WindArr_d, 4*sizeof(int));

	hipMemcpy(proc_seq1_d, proc_seq1, seq_len*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(proc_seq2_d, proc_seq2, seq_len*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(W_d, W, 4*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(WindArr_d, WindArr, 4*sizeof(int), hipMemcpyHostToDevice);

	fillDict<<<1,seq_len>>>(proc_seq1_d,proc_seq2_d, W_d, WindArr_d, min_max_flag, cudaWeight_p_d);
	
	hipMemcpy(proc_seq2, proc_seq2_d, seq_len*sizeof(char), hipMemcpyDeviceToHost);
	hipMemcpy(&cudaWeight, cudaWeight_p_d, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(proc_seq1_d);
	hipFree(proc_seq2_d);
	hipFree(cudaWeight_p_d);
	hipFree(W_d);
	hipFree(WindArr_d);
	
   	return cudaWeight;
}
